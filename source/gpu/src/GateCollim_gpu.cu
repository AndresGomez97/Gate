#include "hip/hip_runtime.h"
#include "GateGPUParticle.hh"
#include "GateToGPUImageSPECT.hh"

#include <dlfcn.h>
#include <stdlib.h>
#include <unistd.h>
#include <pthread.h>

#include "julia.h"

pthread_t tid; 
pthread_mutex_t lock;

struct ThreadArg{
    GateGPUCollimator *collimator;
    GateGPUParticle *particle;
    int *hole;
};

struct RetProjection{
    jl_array_t *px; 
    jl_array_t *py; 
    jl_array_t *pz;
};

__device__ float vector_dot(float3 u, float3 v) {
    return u.x*v.x + u.y*v.y + u.z*v.z;
}

__device__ float3 vector_sub(float3 u, float3 v) {
    return make_float3(u.x-v.x, u.y-v.y, u.z-v.z);
}

__device__ float3 vector_add(float3 u, float3 v) {
    return make_float3(u.x+v.x, u.y+v.y, u.z+v.z);
}

__device__ float3 vector_mag(float3 u, float a) {
    return make_float3(u.x*a, u.y*a, u.z*a);
}

__device__ unsigned int binary_search(float position, float *tab, unsigned int maxid ) {

    unsigned short int begIdx = 0;
    unsigned short int endIdx = maxid - 1;
    unsigned short int medIdx = endIdx / 2;

    while (endIdx-begIdx > 1) {
        if (position < tab[medIdx]) {begIdx = medIdx;}
        else {endIdx = medIdx;}
        medIdx = (begIdx+endIdx) / 2;
    }
    return medIdx;
}

extern "C" {
    __global__ void kernel_map_entry(float *d_px, float *d_py, float *d_pz, 
                                    float *d_entry_collim_y, float *d_entry_collim_z,
                                    int *d_hole, unsigned int y_size, unsigned int z_size,
                                    int particle_size) {
        
        unsigned int id = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
        if (id >= particle_size) {return;}
        if( d_py[ id ] > d_entry_collim_y[ 0 ] || d_py[ id ] < d_entry_collim_y[ y_size - 1 ] )
        {
            d_hole[ id ]=-1;
            return;
        }
            if( d_pz[ id ] > d_entry_collim_z[ 0 ] || d_pz[ id ] < d_entry_collim_z[ z_size - 1 ] )
        {
            d_hole[ id ] = -1;
            return;
        }

        unsigned int index_entry_y = binary_search( d_py[ id ], d_entry_collim_y, y_size );
        unsigned int index_entry_z = binary_search( d_pz[ id ], d_entry_collim_z, z_size );

        unsigned char is_in_hole_y = ( index_entry_y & 1 ) ? 0 : 1;
        unsigned char is_in_hole_z = ( index_entry_z & 1 ) ? 0 : 1;

        unsigned char in_hole = is_in_hole_y & is_in_hole_z;

        d_hole[ id ] = ( in_hole )? index_entry_y * z_size + index_entry_z : -1;
    }
}
extern "C" {
    __global__ void kernel_map_projection(float *d_px, float *d_py, float *d_pz,
                                      float *d_dx, float *d_dy, float *d_dz,
                                      int *d_hole, float planeToProject, 
                                      unsigned int particle_size) {
    
        unsigned int id = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x;
        if( id >= particle_size ) return;
        if( d_hole[ id ] == -1 ) return;

        float3 n  = make_float3( -1.0f, 0.0f, 0.0f );
        float3 v0 = make_float3( planeToProject, 0.0f, 0.0f );
        float3 d  = make_float3( d_dx[ id ], d_dy[ id ], d_dz[ id ] );
        float3 p  = make_float3( d_px[ id ], d_py[ id ], d_pz[ id ] );

        float s = __fdividef( vector_dot( n, vector_sub( v0, p ) ), vector_dot( n, d ) );
        float3 newp = vector_add( p, vector_mag( d, s ) );

        d_px[id] = newp.x;
        d_py[id] = newp.y;
        d_pz[id] = newp.z;
    }
}

extern "C" {
    __global__ void kernel_map_exit(float *d_px, float *d_py, float *d_pz,
                                    float *d_exit_collim_y, float *d_exit_collim_z,
                                    int *d_hole, unsigned int y_size, unsigned int z_size,
                                    int particle_size) {
        
        unsigned int id = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x;
        if( id >= particle_size ) return;
        if( d_hole[ id ] == -1 ) return;

        if( d_py[ id ] > d_exit_collim_y[ 0 ] || d_py[ id ] < d_exit_collim_y[ y_size - 1 ] )
        {
            d_hole[ id ]=-1;
            return;
        }
        if( d_pz[ id ] > d_exit_collim_z[ 0 ] || d_pz[ id ] < d_exit_collim_z[ z_size - 1 ] )
        {
            d_hole[ id ] = -1;
            return;
        }

        unsigned int index_exit_y = binary_search( d_py[ id ], d_exit_collim_y, y_size );
        unsigned int index_exit_z = binary_search( d_pz[ id ], d_exit_collim_z, z_size );

        unsigned char is_in_hole_y = ( index_exit_y & 1 )? 0 : 1;
        unsigned char is_in_hole_z = ( index_exit_z & 1 )? 0 : 1;

        unsigned char in_hole = is_in_hole_y & is_in_hole_z;

        int newhole = ( in_hole )? index_exit_y * z_size + index_exit_z : -1;

        if( newhole == -1 )
        {
            d_hole[ id ] = -1;
            return;
        }

        if( newhole != d_hole[ id ] )
        {
            d_hole[ id ] = -1;
        }
    }
}

void* juliaTask(void *arg){

    pthread_mutex_lock(&lock);

    struct ThreadArg *t_arg = (struct ThreadArg*)arg;
    
    GateGPUCollimator *tcollimator = t_arg->collimator;
    GateGPUParticle *tparticle = t_arg->particle;

    // Read collimator geometry
    unsigned int y_size     = tcollimator->y_size;
    unsigned int z_size     = tcollimator->z_size;
    float planeToProject    = tcollimator->planeToProject + tparticle->px[0];

    // Particles allocation to the Device
    int particle_size = tparticle-> size;

    // Kernel vars
    int block_size = 512;
    int grid_size = (particle_size + block_size - 1) / block_size;
    
    // HANDLER FOR DLOPEN
    void *handle;

    // Julia basic embedding
    typedef void (*t_jl_init)(void);
    typedef jl_value_t *(*t_jl_eval_string)(const char*);
    typedef int (*t_jl_atexit_hook)(int);

    // Arrays
    typedef jl_value_t *(*t_jl_apply_array_type)(jl_value_t*,size_t);
    typedef jl_array_t *(*t_jl_ptr_to_array)(jl_value_t*, void*,size_t,int);

    // Calling Julia methods
    typedef jl_value_t *(*t_jl_get_global)(jl_module_t*, jl_sym_t*);
    typedef jl_value_t *(*t_jl_call)(jl_function_t*, jl_value_t**,int32_t);
    typedef jl_value_t *(*t_jl_call0)(jl_function_t*);
    typedef jl_sym_t *(*t_jl_symbol)(const char*);

    // Box and Unbox types
    typedef jl_value_t *(*t_jl_box_uint32)(uint32_t);
    typedef jl_value_t *(*t_jl_box_int32)(int32_t);
    typedef jl_value_t *(*t_jl_box_int64)(int64_t);
    typedef int64_t (*t_jl_unbox_int64)(jl_value_t*);
    typedef jl_value_t *(*t_jl_box_float32)(float);

    //push and pop GC
    typedef jl_ptls_t (*t_jl_get_ptls_states)(void);

    // LIBJULIA
    handle = dlopen("/home/agmez/julia-1.3.1/lib/libjulia.so", RTLD_LAZY | RTLD_GLOBAL);
    if (!handle) {
    fprintf(stderr, "%s\n", dlerror());
    exit(EXIT_FAILURE);
    }

    dlerror();

    //CASTING METHODS FROM LIBJULIA
    t_jl_init jl_init = (t_jl_init)dlsym(handle, "jl_init__threading");
    t_jl_atexit_hook jl_atexit_hook= (t_jl_atexit_hook)dlsym(handle, "jl_atexit_hook");
    t_jl_eval_string jl_eval_string = (t_jl_eval_string)dlsym(handle, "jl_eval_string");

    t_jl_apply_array_type jl_apply_array_type = (t_jl_apply_array_type)dlsym(handle,"jl_apply_array_type");
    t_jl_ptr_to_array jl_ptr_to_array_1d = (t_jl_ptr_to_array)dlsym(handle,"jl_ptr_to_array_1d");

    t_jl_get_global jl_get_global = (t_jl_get_global)dlsym(handle,"jl_get_global");
    t_jl_call jl_call = (t_jl_call)dlsym(handle,"jl_call");
    t_jl_call0 jl_call0 = (t_jl_call0)dlsym(handle,"jl_call0");
    t_jl_symbol jl_symbol = (t_jl_symbol)dlsym(handle,"jl_symbol");

    t_jl_box_uint32 jl_box_uint32 = (t_jl_box_uint32)dlsym(handle,"jl_box_uint32");
    t_jl_box_int32 jl_box_int32 = (t_jl_box_int32)dlsym(handle,"jl_box_int32");
    t_jl_box_int64 jl_box_int64 = (t_jl_box_int64)dlsym(handle,"jl_box_int64");
    t_jl_box_float32 jl_box_float32 = (t_jl_box_float32)dlsym(handle,"jl_box_float32");
    t_jl_unbox_int64 jl_unbox_int64 = (t_jl_unbox_int64)dlsym(handle,"jl_unbox_int64");

    t_jl_get_ptls_states jl_get_ptls_states = (t_jl_get_ptls_states)dlsym(handle,"jl_get_ptls_states");

    jl_init();
    
    jl_datatype_t *jl_float32_type = *(jl_datatype_t **)dlsym(handle, "jl_float32_type");
    jl_datatype_t *jl_int32_type = *(jl_datatype_t **)dlsym(handle,"jl_int32_type");

    // Include GateKernels.jl code
    jl_eval_string("include(\"/home/agmez/gate/Gate/source/julia/jl/GateKernels.jl\")");

    // Array types for wrappers
    jl_value_t *array_float32 = jl_apply_array_type((jl_value_t*)jl_float32_type, 1);
    jl_value_t *array_int32 = jl_apply_array_type((jl_value_t*)jl_int32_type, 1);
    
    // Module
    jl_module_t *gateKernelsModule = (jl_module_t*)jl_eval_string("GateKernels");
    
    // f_kernel_map_entry, f_kernel_map_projection and f_kernel_map_exit from Module GateKernels
    JL_GC_PUSH1(&gateKernelsModule);
    jl_function_t *f_kernel_map_entry = (jl_function_t*)jl_get_global(gateKernelsModule, jl_symbol("f_kernel_map_entry"));
    jl_function_t *f_kernel_map_projection = (jl_function_t*)jl_get_global(gateKernelsModule,jl_symbol("f_kernel_map_projection"));
    jl_function_t *f_kernel_map_exit = (jl_function_t*)jl_get_global(gateKernelsModule,jl_symbol("f_kernel_map_exit"));
    JL_GC_POP();

    // Wrappers
    // px, py, pz
    jl_array_t *jl_px = jl_ptr_to_array_1d(array_float32, tparticle->px, particle_size, 0);
    jl_array_t *jl_py = jl_ptr_to_array_1d(array_float32, tparticle->py, particle_size, 0);
    jl_array_t *jl_pz = jl_ptr_to_array_1d(array_float32, tparticle->pz, particle_size, 0);

    // entry_collim_y entry_collim_z
    jl_array_t *jl_entry_collim_y = jl_ptr_to_array_1d(array_float32, tcollimator->entry_collim_y, y_size, 0);
    jl_array_t *jl_entry_collim_z = jl_ptr_to_array_1d(array_float32, tcollimator->entry_collim_z, z_size, 0);
    
    // hole
    jl_array_t *jl_hole = jl_ptr_to_array_1d(array_int32, t_arg->hole, particle_size, 0);

    // Args f_kernel_map_entry
    jl_value_t **args;
    JL_GC_PUSHARGS(args,11);
    args[0] = (jl_value_t*)jl_px;
    args[1] = (jl_value_t*)jl_py;
    args[2] = (jl_value_t*)jl_pz;
    args[3] = (jl_value_t*)jl_entry_collim_y;
    args[4] = (jl_value_t*)jl_entry_collim_z;
    args[5] = (jl_value_t*)jl_hole;
    args[6] = jl_box_uint32(y_size);
    args[7] = jl_box_uint32(z_size);
    args[8] = jl_box_int32(particle_size);
    args[9] = jl_box_int32(grid_size);
    args[10] = jl_box_int32(block_size);

    // Call f_kernel_map_entry
    jl_call(f_kernel_map_entry,args,11);
    jl_hole = (jl_array_t *)jl_call(f_kernel_map_entry,args,11);

    // dx, dy, dz
    jl_array_t *jl_dx = jl_ptr_to_array_1d(array_float32, tparticle->dx, particle_size, 0);
    jl_array_t *jl_dy = jl_ptr_to_array_1d(array_float32, tparticle->dy, particle_size, 0);
    jl_array_t *jl_dz = jl_ptr_to_array_1d(array_float32, tparticle->dz, particle_size, 0);

    // Args f_kernel_map_projection
    args[0] = (jl_value_t*)jl_px;
    args[1] = (jl_value_t*)jl_py;
    args[2] = (jl_value_t*)jl_pz;
    args[3] = (jl_value_t*)jl_dx;
    args[4] = (jl_value_t*)jl_dy;
    args[5] = (jl_value_t*)jl_dz;
    args[6] = (jl_value_t*)jl_hole;
    args[7] = jl_box_float32(planeToProject);
    args[8] = jl_box_uint32(particle_size);
    args[9] = jl_box_int32(grid_size);
    args[10] = jl_box_int32(block_size);

    // Call f_kernel_map_projection
    struct RetProjection *retproj = (RetProjection *)jl_call(f_kernel_map_projection,args,11);
    jl_px = retproj->px;
    jl_py = retproj->py;
    jl_pz = retproj->pz;

    // Accessing result data
    tparticle->px = (float *)jl_array_data(jl_px);
    tparticle->py = (float *)jl_array_data(jl_py);
    tparticle->pz = (float *)jl_array_data(jl_pz);
    
    // exit_collim_y exit_collim_z
    jl_array_t *jl_exit_collim_y = jl_ptr_to_array_1d(array_float32, tcollimator->exit_collim_y, y_size, 0);
    jl_array_t *jl_exit_collim_z = jl_ptr_to_array_1d(array_float32, tcollimator->exit_collim_z, z_size, 0);

    // Args f_kernel_map_exit
    args[0] = (jl_value_t*)jl_px;
    args[1] = (jl_value_t*)jl_py;
    args[2] = (jl_value_t*)jl_pz;
    args[3] = (jl_value_t*)jl_exit_collim_y;
    args[4] = (jl_value_t*)jl_exit_collim_z;
    args[5] = (jl_value_t*)jl_hole;
    args[6] = jl_box_uint32(y_size);
    args[7] = jl_box_uint32(z_size);
    args[8] = jl_box_int32(particle_size);
    args[9] = jl_box_int32(grid_size);
    args[10] = jl_box_int32(block_size);

    // Call f_kernel_map_exit
    jl_hole = (jl_array_t*)jl_call(f_kernel_map_exit,args,11);
    JL_GC_POP();

    // Accessing result data
    t_arg->hole = (int*)jl_array_data(jl_hole);
    jl_atexit_hook(0);
    pthread_mutex_unlock(&lock);
	
    return NULL;
}

void GateGPUCollimator_init(GateGPUCollimator *collimator) {

    hipSetDevice(collimator->cudaDeviceID);

    unsigned int y_size = collimator->y_size;
    unsigned int z_size = collimator->z_size;

    unsigned int mem_float_y = y_size * sizeof(float);
    unsigned int mem_float_z = z_size * sizeof(float);
    

    float* d_entry_collim_y;
    float* d_entry_collim_z;
    float* d_exit_collim_y;
    float* d_exit_collim_z;
    
    hipMalloc((void**) &d_entry_collim_y, mem_float_y);
    hipMalloc((void**) &d_entry_collim_z, mem_float_z);
    hipMalloc((void**) &d_exit_collim_y, mem_float_y);
    hipMalloc((void**) &d_exit_collim_z, mem_float_z);

    hipMemcpy(d_entry_collim_y, collimator->entry_collim_y, mem_float_y, hipMemcpyHostToDevice);
    hipMemcpy(d_entry_collim_z, collimator->entry_collim_z, mem_float_z, hipMemcpyHostToDevice);
    hipMemcpy(d_exit_collim_y, collimator->exit_collim_y, mem_float_y, hipMemcpyHostToDevice);
    hipMemcpy(d_exit_collim_z, collimator->exit_collim_z, mem_float_z, hipMemcpyHostToDevice);

    collimator->gpu_entry_collim_y = d_entry_collim_y;
    collimator->gpu_entry_collim_z = d_entry_collim_z;
    collimator->gpu_exit_collim_y = d_exit_collim_y;
    collimator->gpu_exit_collim_z = d_exit_collim_z;
    
}

void GateGPUCollimator_process(GateGPUCollimator *collimator, GateGPUParticle *particle) {
/*    
    hipSetDevice(collimator->cudaDeviceID);
   
    // Read collimator geometry
    float* d_entry_collim_y = collimator->gpu_entry_collim_y;
    float* d_entry_collim_z = collimator->gpu_entry_collim_z;
    float* d_exit_collim_y  = collimator->gpu_exit_collim_y;
    float* d_exit_collim_z  = collimator->gpu_exit_collim_z; 
    unsigned int y_size     = collimator->y_size;
    unsigned int z_size     = collimator->z_size;
    float planeToProject    = collimator->planeToProject + particle->px[0];
*/
    // Particles allocation to the Device
    int particle_size = particle-> size;
    unsigned int mem_float_particle = particle_size * sizeof(float);
    unsigned int mem_int_hole = particle_size * sizeof(int);
/*    float *d_px, *d_py, *d_pz;
    float *d_dx, *d_dy, *d_dz;
    int *d_hole;
    hipMalloc((void**) &d_px, mem_float_particle);
    hipMalloc((void**) &d_py, mem_float_particle);
    hipMalloc((void**) &d_pz, mem_float_particle);
    hipMalloc((void**) &d_dx, mem_float_particle);
    hipMalloc((void**) &d_dy, mem_float_particle);
    hipMalloc((void**) &d_dz, mem_float_particle);
    hipMalloc((void**) &d_hole, mem_int_hole);
*/
    // Array of holes :)
    int *h_hole = (int*)malloc(mem_int_hole);

    // Kernel vars
    //dim3 threads, grid;
    int block_size = 512;
    int grid_size = (particle_size + block_size - 1) / block_size;
    //threads.x = block_size;
    //grid.x = grid_size;
/*
    // Kernel map entry
    kernel_map_entry<<<grid, threads>>>(d_px, d_py, d_pz, 
                                        d_entry_collim_y, d_entry_collim_z,
                                        d_hole, y_size, z_size,
                                        particle_size);

    // Kernel projection
    kernel_map_projection<<<grid, threads>>>(d_px, d_py, d_pz,
                                             d_dx, d_dy, d_dz,
                                             d_hole, planeToProject, particle_size);

    // Kernel map_exit
    kernel_map_exit<<<grid, threads>>>(d_px, d_py, d_pz,
                                       d_exit_collim_y, d_exit_collim_z,
                                       d_hole, y_size, z_size,
                                       particle_size);
    
    // Copy particles from device to host
    hipMemcpy(particle->px, d_px, mem_float_particle, hipMemcpyDeviceToHost);
    hipMemcpy(particle->py, d_py, mem_float_particle, hipMemcpyDeviceToHost);
    hipMemcpy(particle->pz, d_pz, mem_float_particle, hipMemcpyDeviceToHost);
    hipMemcpy(h_hole, d_hole, mem_int_hole, hipMemcpyDeviceToHost);
*/
    struct ThreadArg targ;
    targ.collimator = collimator;
    targ.particle = particle;
    targ.hole = h_hole;

    struct ThreadArg *p_targ = (struct ThreadArg *)malloc(sizeof(struct ThreadArg));
    p_targ=&targ;

    // Julia Task
    if (pthread_mutex_init(&lock, NULL) != 0) { 
        printf("mutex init has failed\n");  
    }
    if (pthread_create(&tid,NULL,&juliaTask,(void *)p_targ) != 0){
        printf("Thread can't be created\n");
    }
    pthread_join(tid,NULL);
    pthread_mutex_destroy(&lock);

    h_hole = targ.hole;

    // Pack data to CPU
    int c = 0;
    int i = 0;
    while( i < particle_size )
    {
        if( h_hole[ i ] == -1 )
        {
            ++i;
            continue;
        }
        //h_hole[ c ] = h_hole[ i ];
        particle->px[ c ] = particle->px[ i ];
        particle->py[ c ] = particle->py[ i ];
        particle->pz[ c ] = particle->pz[ i ];
        particle->dx[ c ] = particle->dx[ i ];
        particle->dy[ c ] = particle->dy[ i ];
        particle->dz[ c ] = particle->dz[ i ];
				particle->eventID[ c ] = particle->eventID[ i ];
				particle->parentID[ c ] = particle->parentID[ i ];
				particle->trackID[ c ] = particle->trackID[ i ];
				particle->t[ c ] = particle->t[ i ];
				particle->E[ c ] = particle->E[ i ];
				particle->type[ c ] = particle->type[ i ];
        ++c;
        ++i;
    }
    particle->size = c;    
/*
    // Free memory
    hipFree(d_px);
    hipFree(d_py);
    hipFree(d_pz);
    hipFree(d_dx);
    hipFree(d_dy);
    hipFree(d_dz);
    hipFree(d_hole);
*/
    free(h_hole);
}